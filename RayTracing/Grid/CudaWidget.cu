#include "hip/hip_runtime.h"
#include "CudaWidget.h"


CudaWidget::CudaWidget(uchar4 ** dev , uchar * d) 
{
	setMinimumSize(640,480);
	QLayout * main = new QHBoxLayout();
	setLayout(main);
	windowWidth = width();
	windowHeight = height();
	size = windowWidth * windowHeight * 4;
	readBuffer = d;
	QImage r(readBuffer,windowWidth,windowHeight,QImage::Format::Format_RGBA8888);
	readImage = r;

	devReadBuffer = dev;
	connect(&updateTimer,&QTimer::timeout,this, &CudaWidget::update);
	updateTimer.start();

	main->addWidget(&imageLabel);
}


CudaWidget::~CudaWidget(void)
{
	//delete [] readBuffer;
	//release(*devReadBuffer);
}


void CudaWidget::update(void)
{
	timer.start();
	draw(640,480,*devReadBuffer);
	copyImage(*devReadBuffer, readBuffer,size);

	imageLabel.setPixmap(QPixmap::fromImage(readImage));
	float interval = (timer.elapsed()/1000.0f);
	float fps = 1.0f/interval;
	setWindowTitle(QString::number(fps));
}