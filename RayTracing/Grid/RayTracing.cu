#include "hip/hip_runtime.h"
#include <iostream> 
#include <vector> 
#include <hip/hip_runtime.h> 
#include <hip/hip_runtime.h>
#include <thrust\detail\mpl\math.h>
#include <math_functions.h>
#include <common_functions.h>
#include "hip/hip_vector_types.h"
#include <QtWidgets\qwidget.h>
#include <QtWidgets\qapplication.h>
#include <QtCore\qtimer.h>
#include "RenderWindow.h"
#include "MainWindow.h"
#include "CudaWidget.h"

#pragma comment(lib, "cudart") 


using std::cerr; 
using std::cout; 
using std::endl; 
using std::exception; 
using std::vector; 


static void HandleError( hipError_t err,
						const char *file,
						int line ) {
							if (err != hipSuccess) {
								printf( "%s in %s at line %d\n", hipGetErrorString( err ),
									file, line );
								system("pause");
								exit( EXIT_FAILURE );
							}
}
#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))
#define INF 10000000

struct CSphere
{
	float3 rgb;
	float3 position;
	float radius;

	__device__ float hit(float2 pixel, float * n)
	{
		float2 delta = pixel - make_float2(position.x,position.y);
		if(dot(delta, delta) < radius * radius)
		{
			float dz = sqrtf(radius * radius - (delta.x * delta.x - delta.y * delta.y));
			return dz + position.z;
		}

		return -INF;
	}
};


__global__ void colorKernel(uchar4 * image)
{
	int x = blockIdx.x;
	int y = blockIdx.y;

	int offset = x + y * gridDim.x;
	//image[offset] =make_uchar4( 100,0,0,255);
	image[offset].x = 100;//make_uchar4( 100,0,0,255);
	image[offset].w = 255;//make_uchar4( 100,0,0,255);

}


void intitializeDev(uchar4 ** dev , int size)
{	
	HANDLE_ERROR(hipMalloc(dev, size));
}


void copyImage(uchar4 * dev, unsigned char * dst, int size)
{
	//	HANDLE_ERROR(hipMemcpy(dst, dev,size, hipMemcpyDeviceToHost));
	HANDLE_ERROR(hipMemcpy(dst, dev,size, hipMemcpyDeviceToHost));
}


void draw(int width, int height , uchar4 * dev)
{
	dim3 grid(width, height,1);
	colorKernel<<< grid , 1 >>>(dev);
}

void release(uchar4 * dev)
{
	hipFree(dev);
}


void update()
{

}



//class CudaWidget : public QWidget
//{
//public:
//	CudaWidget(uchar4 ** dev  , uchar * d) 
//	{
//		setMinimumSize(640,480);
//		QLayout * main = new QHBoxLayout();
//		setLayout(main);
//		windowWidth = width();
//		windowHeight = height();
//		size = windowWidth * windowHeight * 4;
//		readBuffer = d;
//		QImage r(readBuffer,windowWidth,windowHeight,QImage::Format::Format_RGBA8888);
//		readImage = r;
//
//		devReadBuffer = dev;
//		connect(&updateTimer,&QTimer::timeout,this, &CudaWidget::update);
//		updateTimer.start();
//
//		main->addWidget(&imageLabel);
//	}
//
//
//	~CudaWidget(void)
//	{
//		//delete [] readBuffer;
//		//release(*devReadBuffer);
//	}
//
//
//	int windowWidth;
//	int windowHeight;
//	float3 color;
//	QImage readImage;
//	uchar * readBuffer;
//	uchar4 ** devReadBuffer;
//	QLabel imageLabel;
//	QTimer updateTimer;
//	int size;
//
//	void update(void)
//	{
//
//		draw(640,480,*devReadBuffer);
//		copyImage(*devReadBuffer, readBuffer,size);
//
//		imageLabel.setPixmap(QPixmap::fromImage(QImage(readBuffer,640,480,QImage::Format::Format_RGBA8888)));
//	}
//
//
//};
#include <hip/hip_texture_types.h>
#include <texture_types.h>

//int main(int argc, char * argv []) 
//{
//
//	QApplication app(argc,argv);
//	int size  = 640 * 480 * 4;
//	unsigned char * d = new unsigned char[(size)];
//	uchar4 * dev = nullptr;
//	HANDLE_ERROR(hipMalloc(&dev, size));
//	//intitializeDev(&dev,size);
//	draw(640,480,dev);
//	//copyImage(dev, d,size);
//	text
//	HANDLE_ERROR(hipMemcpy(d, dev,size, hipMemcpyDeviceToHost));
//
//	//QLabel c;
//	//c.setPixmap(QPixmap::fromImage(QImage(d,640,480,QImage::Format::Format_RGBA8888)));
//
//	CudaWidget c(&dev, d);
//	c.show();
//	//QLabel cl;
//	//cl.setPixmap(QPixmap::fromImage(QImage(d,640,480,QImage::Format::Format_RGBA8888)));
//	//cl.show();
//	return app.exec();
//}